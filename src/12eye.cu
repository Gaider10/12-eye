#include "hip/hip_runtime.h"
#include "xrsr.h"
#include "skip.cuh"
#include <cstdint>
#include <cinttypes>
#include <cstring>
#include <cstdio>
#include <time.h>
#include <vector>
#include <thread>
#include <random>
#include <stdexcept>

#include "lib.h"

constexpr uint32_t threads_per_block = 256;

using InputChunkPos = Layout;
constexpr uint32_t inputs_size = 1 << 16;
__managed__ InputChunkPos inputs[inputs_size];
__managed__ uint32_t inputs_count;

struct OutputChunkPos {
    uint64_t world_seed;
    int16_t start_chunk_x;
    int16_t portal_chunk_x;
    int16_t start_chunk_z;
    int16_t portal_chunk_z;
};
__managed__ OutputChunkPos outputs[4096];
__managed__ uint64_t outputs_count;

struct PrecompItem {
    // XRSR128 xrsr;
    uint32_t xrsr[4];
    // uint32_t pad;
};
constexpr uint32_t precomp_size = 32 * 20;
__device__ PrecompItem precomp_global[precomp_size];

__device__ void xrsr128_xor(XRSR128 *rng, XRSR128 xor_) {
    rng->lo ^= xor_.lo;
    rng->hi ^= xor_.hi;
}

__device__ void xrsr128_xor(XRSR128 *rng, PrecompItem &xor_) {
    rng->lo ^= ((uint64_t)xor_.xrsr[1] << 32) | xor_.xrsr[0];
    rng->hi ^= ((uint64_t)xor_.xrsr[3] << 32) | xor_.xrsr[2];
}

__device__ int32_t xrsr128_next_bits(XRSR128 *rng, int32_t bits) {
    return (int32_t)((int64_t)xrsr_long(rng) >> (64 - bits));
}

__device__ int64_t xrsr128_nextLong(XRSR128 *rng) {
    return ((int64_t)xrsr128_next_bits(rng, 32) << 32) + (int64_t)xrsr128_next_bits(rng, 32);
}

__device__ void xrsr128_setFeatureSeed(XRSR128 *rng, uint64_t world_seed, int32_t x, int32_t z, int32_t index, int32_t step) {
    xrsr_seed(rng, world_seed);
    int64_t a = xrsr128_nextLong(rng) | 1LL;
    int64_t b = xrsr128_nextLong(rng) | 1LL;
    int64_t decorationSeed = (int64_t)x * a + (int64_t)z * b ^ world_seed;
    int64_t featureSeed = decorationSeed + (int64_t)index + (int64_t)(10000 * step);
    xrsr_seed(rng, featureSeed);
}

__global__ void filter() {
    __shared__ PrecompItem precomp_shared[precomp_size];
    constexpr uint32_t precomp_size_u32 = precomp_size * sizeof(precomp_global[0]) / sizeof(uint32_t);
    for (uint32_t i = 0; i < precomp_size_u32 / threads_per_block; i++) {
        uint32_t index = i * threads_per_block + threadIdx.x;
        reinterpret_cast<uint32_t*>(precomp_shared)[index] = reinterpret_cast<uint32_t*>(precomp_global)[index];
    }
    if (precomp_size_u32 % threads_per_block != 0 && threadIdx.x < precomp_size_u32 % threads_per_block) {
        uint32_t index = precomp_size_u32 / threads_per_block * threads_per_block + threadIdx.x;
        reinterpret_cast<uint32_t*>(precomp_shared)[index] = reinterpret_cast<uint32_t*>(precomp_global)[index];
    }
    __syncthreads();

    uint64_t index = (uint64_t)blockIdx.x * blockDim.x + (uint64_t)threadIdx.x;
    uint64_t world_seed_hi = index & 0xFFFF;
    uint64_t input_chunk_index = index >> 16;
    if (input_chunk_index >= inputs_count) return;

    InputChunkPos inputChunkPos = inputs[input_chunk_index];
    uint64_t world_seed = (world_seed_hi << 48) | inputChunkPos.structure_seed;
    XRSR128 rng;
    xrsr128_setFeatureSeed(&rng, world_seed, (int32_t)inputChunkPos.portal_chunk_x << 4, (int32_t)inputChunkPos.portal_chunk_z << 4, 19, 4);
    uint64_t lo = rng.lo;
    uint64_t hi = rng.hi;

    skip_gpu(&rng);

    xrsr128_xor(&rng, precomp_shared[32 *  0 + ((lo >>       0) & 31)]);
    xrsr128_xor(&rng, precomp_shared[32 *  1 + ((lo >>       5) & 31)]);
    xrsr128_xor(&rng, precomp_shared[32 *  2 + ((lo >>      10) & 31)]);
    xrsr128_xor(&rng, precomp_shared[32 *  3 + ((lo >>      15) & 31)]);
    xrsr128_xor(&rng, precomp_shared[32 *  4 + ((lo >>      20) & 31)]);
    xrsr128_xor(&rng, precomp_shared[32 *  5 + ((lo >>      25) & 31)]);
    xrsr128_xor(&rng, precomp_shared[32 *  6 + ((lo >> 32 +  0) & 31)]);
    xrsr128_xor(&rng, precomp_shared[32 *  7 + ((lo >> 32 +  5) & 31)]);
    xrsr128_xor(&rng, precomp_shared[32 *  8 + ((lo >> 32 + 10) & 31)]);
    xrsr128_xor(&rng, precomp_shared[32 *  9 + ((lo >> 32 + 15) & 31)]);
    xrsr128_xor(&rng, precomp_shared[32 * 10 + ((lo >> 32 + 20) & 31)]);
    xrsr128_xor(&rng, precomp_shared[32 * 11 + ((lo >> 32 + 25) & 31)]);
    xrsr128_xor(&rng, precomp_shared[32 * 12 + ((hi >>       0) & 31)]);
    xrsr128_xor(&rng, precomp_shared[32 * 13 + ((hi >>       5) & 31)]);
    xrsr128_xor(&rng, precomp_shared[32 * 14 + ((hi >>      10) & 31)]);
    xrsr128_xor(&rng, precomp_shared[32 * 15 + ((hi >>      15) & 31)]);
    xrsr128_xor(&rng, precomp_shared[32 * 16 + ((hi >>      20) & 31)]);
    xrsr128_xor(&rng, precomp_shared[32 * 17 + ((hi >>      25) & 31)]);
    xrsr128_xor(&rng, precomp_shared[32 * 18 + ((hi >> 32 +  0) & 31)]);
    xrsr128_xor(&rng, precomp_shared[32 * 19 + ((hi >> 32 +  5) & 31)]);

    for (int j = 0; j < 12; j++) {
        if (xrsr_long(&rng) < 16602070326045573120ULL) {
            return;
        }
    }

    outputs[atomicAdd((unsigned long long *) &outputs_count, 1)] = OutputChunkPos{world_seed, inputChunkPos.start_chunk_x, inputChunkPos.portal_chunk_x, inputChunkPos.start_chunk_z, inputChunkPos.portal_chunk_z};
}

#define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
   if (code != hipSuccess) {
      std::fprintf(stderr, "Cuda Error: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

bool profiling = false;
bool no_layouts = false;

// constexpr uint32_t layout_thread_count = 12;
// constexpr uint32_t thread_inputs_size = inputs_size / layout_thread_count;

// InputChunkPos thread_inputs[layout_thread_count][thread_inputs_size];
// uint64_t thread_inputs_count[layout_thread_count] = {};
// std::thread layout_threads[layout_thread_count];

// void start_layout_threads(uint32_t structure_seed_hi) {
//     uint64_t full_structure_seed_start = (uint64_t)structure_seed_hi << 16;
//     uint64_t full_structure_seed_count = 1 << 16;
//     if (profiling) full_structure_seed_count /= 32;

//     for (uint32_t i = 0; i < layout_thread_count; i++) {
//         layout_threads[i] = std::thread([=](){
//             uint64_t structure_seed_start = full_structure_seed_start + i * full_structure_seed_count / layout_thread_count;
//             uint64_t structure_seed_end = full_structure_seed_start + (i + 1) * full_structure_seed_count / layout_thread_count;
//             thread_inputs_count[i] = generate_layouts(structure_seed_start, structure_seed_end, thread_inputs[i], thread_inputs_size);
//         });
//     }
// }

// void join_layout_threads() {
//     for (int i = 0; i < layout_thread_count; i++) {
//         layout_threads[i].join();
//     }
// }

struct LayoutThreadData {
    std::thread thread;
    std::vector<InputChunkPos> inputs;

    LayoutThreadData(uint32_t inputs_size) : thread(), inputs(inputs_size) {

    }
};

enum class LayoutThreadPoolState {
    Empty,
    Running,
    HasData,
};

struct LayoutThreadPool {
    LayoutThreadPool(uint32_t thread_count) : threads(), state(LayoutThreadPoolState::Empty) {
        uint32_t thread_inputs_size = inputs_size / thread_count;

        threads.reserve(thread_count);
        for (uint32_t i = 0; i < thread_count; i++) {
            threads.emplace_back(thread_inputs_size);
        }
    }

    LayoutThreadPoolState get_state() const {
        return state;
    }

    void start_layout_threads(uint32_t structure_seed_hi) {
        if (state == LayoutThreadPoolState::Running) throw std::runtime_error("Already Running");

        uint64_t full_structure_seed_start = (uint64_t)structure_seed_hi << 16;
        uint64_t full_structure_seed_count = 1 << 16;
        if (profiling) full_structure_seed_count /= 32;

        for (uint32_t i = 0; i < threads.size(); i++) {
            auto &thread_data = threads[i];
            uint64_t structure_seed_start = full_structure_seed_start + i * full_structure_seed_count / threads.size();
            uint64_t structure_seed_end = full_structure_seed_start + (i + 1) * full_structure_seed_count / threads.size();
            auto &thread_inputs = thread_data.inputs;

            thread_data.thread = std::thread([=, &thread_inputs](){
                uint32_t count = generate_layouts(structure_seed_start, structure_seed_end, thread_inputs.data(), thread_inputs.size());
                thread_inputs.resize(count);
            });
        }

        state = LayoutThreadPoolState::Running;
    }

    void join_layout_threads() {
        if (state != LayoutThreadPoolState::Running) throw std::runtime_error("Not Running");

        for (auto &thread_data : threads) {
            thread_data.thread.join();
        }

        state = LayoutThreadPoolState::HasData;
    }

    void copy_data() {
        if (state != LayoutThreadPoolState::HasData) throw std::runtime_error("Not HasData");

        inputs_count = 0;
        for (auto &thread_data : threads) {
            uint32_t count = thread_data.inputs.size();
            cudaCheckError( hipMemcpy(inputs + inputs_count, thread_data.inputs.data(), count * sizeof(inputs[0]), hipMemcpyHostToDevice) );
            inputs_count += count;
        }
    }

private:
    std::vector<LayoutThreadData> threads;
    LayoutThreadPoolState state;
};

void run(uint32_t structure_seed_hi, LayoutThreadPool &layout_thread_pool) {
    outputs_count = 0;
    inputs_count = 0;

    if (layout_thread_pool.get_state() == LayoutThreadPoolState::Empty) {
        layout_thread_pool.start_layout_threads(structure_seed_hi);
        layout_thread_pool.join_layout_threads();
    }

    layout_thread_pool.copy_data();

    // printf("inputs_count = %" PRIu64 " invocations = %" PRIu64 "\n", inputs_count, inputs_count * COUNT16);

    uint32_t thread_count = inputs_count * (1 << 16);
    uint32_t block_count = (thread_count - 1) / threads_per_block + 1;
    filter<<<block_count, threads_per_block>>>();
    cudaCheckError( hipPeekAtLastError() );

    if (!no_layouts) {
        layout_thread_pool.start_layout_threads(structure_seed_hi + 1);
    }

    cudaCheckError( hipDeviceSynchronize() );

    auto start = std::chrono::steady_clock::now();

    if (!no_layouts) {
        layout_thread_pool.join_layout_threads();
    }

    auto end = std::chrono::steady_clock::now();
    double elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count() * 1E-9;
    if (elapsed > 0.005) {
        // printf("CPU Thread join took %.3f s\n", elapsed);
    }

    for (uint64_t i = 0; i < outputs_count; i++) {
        OutputChunkPos outputChunkPos = outputs[i];
        // InputChunkPos inputChunkPos = inputs[outputChunkPos.input_index];
        // int64_t world_seed = ((uint64_t)outputChunkPos.world_seed_hi << 48) | ((uint64_t)inputChunkPos.structure_seed_hi << 32) | structure_seed_lo;
        // int64_t world_seed = ((uint64_t)outputChunkPos.world_seed_hi << 48) | inputChunkPos.structure_seed;
        // int startChunkX = 0;
        // int startChunkZ = 0;
        // stronghold_generator::StrongholdGenerator::getFirstPosFast(world_seed, startChunkX, startChunkZ);
        // printf("Seed: %lli Start: %i %i Pos: %i ~ %i\n", world_seed, startChunkX, startChunkZ, (int32_t)inputChunkPos.chunk_x << 4, (int32_t)inputChunkPos.chunk_z << 4);
        bool is_valid = test_world_seed(outputChunkPos.world_seed, outputChunkPos.start_chunk_x, outputChunkPos.start_chunk_z);
        std::printf("Seed: %" PRIi64 " Start: %i %i Pos: %i ~ %i Valid: %s\n", outputChunkPos.world_seed, outputChunkPos.start_chunk_x, outputChunkPos.start_chunk_z, outputChunkPos.portal_chunk_x << 4, outputChunkPos.portal_chunk_z << 4, is_valid ? "YES" : "no");
    }
}

void precompute_bits(PrecompItem *table, unsigned int first, unsigned int count) {
    for (uint64_t bits = 0; bits < ((uint64_t) 1 << count); bits++) {
        uint64_t seed_lo = 0;
        if (first < 64) seed_lo = bits << first;
        uint64_t seed_hi = 0;
        if (first + count > 64) {
            if (first >= 64) seed_hi = bits << (first - 64);
            else seed_hi = bits >> (64 - first);
        }

        XRSR128 rng;
        xrsr128_init(&rng, 0, 0);
        skip_cpu(&rng, seed_lo, seed_hi);
        table[bits] = PrecompItem { { (uint32_t)rng.lo, (uint32_t)(rng.lo >> 32), (uint32_t)rng.hi, (uint32_t)(rng.hi >> 32) } };
    }
}

template<typename T>
void precompute_symbol(T &symbol, uint64_t offset, uint32_t first, uint32_t count) {
    uint32_t table_size = 1 << count;
    PrecompItem *data = new PrecompItem[table_size];
    precompute_bits(data, first, count);
    PrecompItem *address = 0;
    cudaCheckError( hipGetSymbolAddress((void**) &address, symbol) );
    cudaCheckError( hipMemcpy(address + offset, data, table_size * sizeof(data[0]), hipMemcpyHostToDevice) );
    delete[] data;
}

void precompute() {
    auto start = std::chrono::steady_clock::now();

    precompute_symbol(precomp_global, 32 *  0,       0, 5);
    precompute_symbol(precomp_global, 32 *  1,       5, 5);
    precompute_symbol(precomp_global, 32 *  2,      10, 5);
    precompute_symbol(precomp_global, 32 *  3,      15, 5);
    precompute_symbol(precomp_global, 32 *  4,      20, 5);
    precompute_symbol(precomp_global, 32 *  5,      25, 5);
    precompute_symbol(precomp_global, 32 *  6, 32 +  0, 5);
    precompute_symbol(precomp_global, 32 *  7, 32 +  5, 5);
    precompute_symbol(precomp_global, 32 *  8, 32 + 10, 5);
    precompute_symbol(precomp_global, 32 *  9, 32 + 15, 5);
    precompute_symbol(precomp_global, 32 * 10, 32 + 20, 5);
    precompute_symbol(precomp_global, 32 * 11, 32 + 25, 5);
    precompute_symbol(precomp_global, 32 * 12, 64 +  0, 5);
    precompute_symbol(precomp_global, 32 * 13, 64 +  5, 5);
    precompute_symbol(precomp_global, 32 * 14, 64 + 10, 5);
    precompute_symbol(precomp_global, 32 * 15, 64 + 15, 5);
    precompute_symbol(precomp_global, 32 * 16, 64 + 20, 5);
    precompute_symbol(precomp_global, 32 * 17, 64 + 25, 5);
    precompute_symbol(precomp_global, 32 * 18, 96 +  0, 5);
    precompute_symbol(precomp_global, 32 * 19, 96 +  5, 5);

    auto end = std::chrono::steady_clock::now();
    double delta = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count() * 1E-9;
    // std::fprintf(stderr, "Precomputing done in %.2fs\n", delta);
}

void bench_layout() {
    uint32_t out_len = 1 << 16;
    std::vector<Layout> out(out_len);

    auto start = std::chrono::steady_clock::now();

    for (uint64_t i = 0;; i++) {
        uint32_t count = generate_layouts(i * out_len, i * out_len + out_len, out.data(), out_len);
        std::printf("%" PRIu32 " / %" PRIu32 "\n", count, out_len);

        uint64_t print_interval = 1;
        uint64_t new_i = i + 1;
        if (new_i % print_interval == 0) {
            auto end = std::chrono::steady_clock::now();
            double delta = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count() * 1E-9;
            double per_sec = print_interval * out_len / delta;
            std::printf("%" PRIu64 " %.3f s %.3f sps\n", new_i, delta, per_sec);
            start = end;
        }
    }
}

int main(int argc, char **argv) {
    bool bench = false;

    uint32_t start = UINT32_MAX;
    uint32_t end = UINT32_MAX;
    uint32_t count = UINT32_MAX;
    uint32_t print_interval = 8;
    uint32_t threads = 12;

    for (int i = 1; i < argc; i++) {
        if (std::strcmp("--bench", argv[i]) == 0) {
            bench = true;
        } else if (std::strcmp("--profile", argv[i]) == 0) {
            profiling = true;
        } else if (std::strcmp("--no-layouts", argv[i]) == 0) {
            no_layouts = true;
        } else if (std::strcmp("--start", argv[i]) == 0) {
            i += 1;
            if (std::sscanf(argv[i], "%" SCNu32, &start) != 1) {
                std::fprintf(stderr, "Invalid --start: %s\n", argv[i]);
                return 1;
            }
        } else if (std::strcmp("--end", argv[i]) == 0) {
            i += 1;
            if (std::sscanf(argv[i], "%" SCNu32, &end) != 1) {
                std::fprintf(stderr, "Invalid --end: %s\n", argv[i]);
                return 1;
            }
        } else if (std::strcmp("--count", argv[i]) == 0) {
            i += 1;
            if (std::sscanf(argv[i], "%" SCNu32, &count) != 1) {
                std::fprintf(stderr, "Invalid --count: %s\n", argv[i]);
                return 1;
            }
        } else if (std::strcmp("--threads", argv[i]) == 0) {
            i += 1;
            if (std::sscanf(argv[i], "%" SCNu32, &threads) != 1 || threads < 1) {
                std::fprintf(stderr, "Invalid --threads: %s\n", argv[i]);
                return 1;
            }
        } else if (std::strcmp("--print-interval", argv[i]) == 0) {
            i += 1;
            if (std::sscanf(argv[i], "%" SCNu32, &print_interval) != 1) {
                std::fprintf(stderr, "Invalid --print-interval: %s\n", argv[i]);
                return 1;
            }
        } else {
            std::fprintf(stderr, "Unknwon arg: %s\n", argv[i]);
            return 1;
        }
    }

    if (bench) {
        bench_layout();
        return 0;
    }

    if (start == UINT32_MAX) {
        std::random_device rd;
        std::mt19937 rng(rd());
        std::uniform_int_distribution<uint32_t> dist(0, UINT32_MAX);
        start = dist(rng);
    }
    if (count != UINT32_MAX) {
        end = start + count;
    }
    if (end == UINT32_MAX) {
        end = start;
    }

    std::fprintf(stderr, "start = %"  PRIu32 "\n", start);
    std::fprintf(stderr, "end = %"  PRIu32 "\n", end);
    std::fprintf(stderr, "threads = %"  PRIu32 "\n", threads);
    std::fprintf(stderr, "print_interval = %" PRIu32 "\n", print_interval);

    precompute();

    LayoutThreadPool layout_thread_pool(threads);

    // hipDeviceProp_t prop;
    // cudaGetDeviceProperties_v2(&prop, 0);
    // std::printf("prop.sharedMemPerBlock = %zu\n", prop.sharedMemPerBlock);
    // std::printf("prop.sharedMemPerMultiprocessor = %zu\n", prop.sharedMemPerMultiprocessor);
    // std::printf("prop.sharedMemPerBlockOptin = %zu\n", prop.sharedMemPerBlockOptin);

    // cudaCheckError(hipFuncSetAttribute(reinterpret_cast<const void*>(filter), hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxShared));
    // hipFuncAttributes attr;
    // cudaCheckError(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(filter)));
    // std::printf("attr.sharedSizeBytes = %zu\n", attr.sharedSizeBytes);
    // std::printf("attr.constSizeBytes = %zu\n", attr.constSizeBytes);
    // std::printf("attr.localSizeBytes = %zu\n", attr.localSizeBytes);

    auto time_start = std::chrono::steady_clock::now();

    for (uint32_t iter = 0;; iter++) {
        uint32_t structure_seed_hi = start + iter;
        run(structure_seed_hi, layout_thread_pool);
        structure_seed_hi += 1;

        if (print_interval != 0 && (iter + 1) % print_interval == 0) {
            auto time_end = std::chrono::steady_clock::now();
            double delta = std::chrono::duration_cast<std::chrono::nanoseconds>(time_end - time_start).count() * 1E-9;
            uint64_t seeds_per_run = UINT64_C(1) << 32;
            double sps = print_interval * seeds_per_run / delta;
            std::fprintf(stderr, "%" PRIu32 " %.2f Gsps %.2f h\n", structure_seed_hi, sps * 1E-9, (end - structure_seed_hi) * seeds_per_run / sps / 3600);
            time_start = time_end;
        }

        if (structure_seed_hi == end) break;
    }

    return 0;
}